#include "hip/hip_runtime.h"
// Cuda libraries automatically included at compile-time by magic of Cuda toolkit

#include <iostream>

// Kernel function to run on each thread
__global__ void vectorAdd(int* a, int* b, int* c) {
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

int main(void) {
	
	int a[] = {1,2,3,4,5,6};
	int b[] = {2,3,4,5,6,7};
	auto NUMBER_OF_VECTORS = sizeof(a) / sizeof(int);
	int c[NUMBER_OF_VECTORS] = {0};

	// create pointers into the GPU
	int* cudaA;
	int* cudaB;
	int* cudaC;

	// allocate memory in the GPU
	hipMalloc(&cudaA, sizeof(a));
	hipMalloc(&cudaB, sizeof(b));
	hipMalloc(&cudaC, sizeof(c));

	// copy into GPU
	hipMemcpy(cudaA, a, sizeof(a), hipMemcpyHostToDevice);
	hipMemcpy(cudaB, b, sizeof(a), hipMemcpyHostToDevice);

	auto GRID_SIZE = 1; 				 	// number of blocks in grid
	auto BLOCK_SIZE = NUMBER_OF_VECTORS; 	// size of elements in block

	vectorAdd <<< GRID_SIZE, BLOCK_SIZE >>> (cudaA, cudaB, cudaC);

	// copy back out of GPU
	hipMemcpy(c, cudaC, sizeof(c), hipMemcpyDeviceToHost);

	for (int i = 0; i < NUMBER_OF_VECTORS; i++) {
		std::cout << c[i] << " ";
    }
    std::cout << std::endl;

	return 0;
}

