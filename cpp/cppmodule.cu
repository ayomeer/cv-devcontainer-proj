#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cmath>
#include <opencv2/core/cuda/common.hpp>
#include <opencv2/opencv.hpp>
#include <opencv2/cudev.hpp>
#include <pybind11/pybind11.h>
#include <pybind11/numpy.h>
#include <chrono>

namespace py = pybind11;

typedef std::uint8_t imgScalar;
typedef double matScalar;


using namespace std;
using namespace cv;


// === Interface Class ======================================================================

class CppHomography {
public:
    CppHomography() {}
    void setMember(int newVal) { memberVar = newVal; }
    int getMember() { return memberVar; }

    int publicVar;

    void pointwiseUndistort( py::array_t<imgScalar>& pyImg_d, 
                        py::array_t<matScalar>& pyH, 
                        py::tuple img_u_shape,
                        py::array_t<float> pyContour);

private:
    int memberVar;

};




// === Cuda device code (Kernel) ============================================================
__device__ void deviceFcn() {

}

__global__ void undistortKernel
(
    const cv::cuda::PtrStepSz<uchar3> src,
    cv::cuda::PtrStepSz<uchar3> dst,
    double* H,
    cv::InputArray c
)
{
    // Get dst pixel indexes for this thread from CUDA framework
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int j = blockIdx.y * blockDim.y + threadIdx.y;

    /* // Homography Matprod
    // H*xu_hom 
    float xd_hom_0 = H[0]*i + H[1]*j + H[2];
    float xd_hom_1 = H[3]*i + H[4]*j + H[5];
    float xd_hom_2 = H[6]*i + H[7]*j + H[8];

    // Convert to inhom and round to int for use as indexes
    int xd_0 = (int)(xd_hom_0 / xd_hom_2); // x
    int xd_1 = (int)(xd_hom_1 / xd_hom_2); // y

    // Get rgb value from src image 
    dst.ptr(i)[j] = src.ptr(xd_0)[xd_1];
    */
    deviceFcn();

}


void CppHomography::pointwiseUndistort( py::array_t<imgScalar>& pyImg_d, 
                        py::array_t<matScalar>& pyH, 
                        py::tuple img_u_shape,
                        py::array_t<float> pyContour
)
{
    // === Input data preparation ===========================================================  
    // Link pyImg_d data to cv::Mat object img
    Mat img_d(
        pyImg_d.shape(0),               // rows
        pyImg_d.shape(1),               // cols
        CV_8UC3,                        // data type
        (imgScalar*)pyImg_d.data());    // data pointer
    
    // Link pyH data to C-array
    double* arrH = (matScalar*)pyH.data(); // or: const double* arrH = pyH.data();

    // Cast py::tuple into int
    int M = img_u_shape[0].cast<int>();
    int N = img_u_shape[1].cast<int>();

    // Link pyContour to C-array
    //const int* arrC = pyContour.data();
    Mat contour(
        pyContour.shape(0),
        pyContour.shape(1),
        CV_32FC1,
        (float*)pyContour.data()
    );
    cv::InputArray inpContour(contour);

    // ===  CUDA host code ==================================================================

    // Load H-array onto device, so it doesn't have to be passed to each kernel individually
    double* dPtr_H = 0; // device pointer to copy of H on GPU
    hipMalloc(&dPtr_H, pyH.shape(0)*pyH.shape(1)*sizeof(double));
    hipMemcpy(dPtr_H, arrH, pyH.shape(0)*pyH.shape(1)*sizeof(double), hipMemcpyHostToDevice);


    // Prep input and return images
    Mat img;
    cvtColor(img_d, img, COLOR_RGB2BGR);
    
    cv::cuda::GpuMat src;
    
    Mat ret;
    cv::cuda::GpuMat dst(M, N, CV_8UC3); // allocate space for dst image
    
    // Prep kernel launch
    src.upload(img);
    
    const dim3 blockSize(16,16);
    const dim3 gridSize(cv::cudev::divUp(dst.cols, blockSize.x), 
                        cv::cudev::divUp(dst.rows, blockSize.y)); 


    // -- Kernel Launch 1 (slow) ------------------------------------------------------------ 

    src.upload(img);
    
    undistortKernel<<<gridSize, blockSize>>>(src, dst, dPtr_H, inpContour);
    hipDeviceSynchronize();
    
    dst.download(ret);

    // -- Kernel Launch 2 (fast) ----------------------------------------------------------- 
    auto start = chrono::steady_clock::now();
    src.upload(img);
    
    undistortKernel<<<gridSize, blockSize>>>(src, dst, dPtr_H, inpContour);
    hipDeviceSynchronize();
    
    dst.download(ret);
    auto end = chrono::steady_clock::now();

    // -------------------------------------------------------------------------------------

    // show results
    cout << "tKernel: "
        << chrono::duration_cast<chrono::microseconds>(end - start).count()
        << " µs" << endl;

    imshow("ret image", ret);
    waitKey(0);


    return;
}       

PYBIND11_MODULE(cppmodule, m){
    m.doc() = "Docstring for cpp homography module";
    // m.def("pointwiseUndistort", &pointwiseUndistort, py::return_value_policy::automatic);

    py::class_<CppHomography>(m, "CppHomography")
        .def(py::init()) // Wrap class constructor
        .def("setMember", &CppHomography::setMember)
        .def("getMember", &CppHomography::getMember)
        .def("pointwiseUndistort", &CppHomography::pointwiseUndistort)
        .def_readwrite("publicVar", &CppHomography::publicVar)
        ;

    




    // Returning Mat to Python as Numpy array
    py::class_<cv::Mat>(m, "Mat", py::buffer_protocol()) 
        .def_buffer([](cv::Mat &im) -> py::buffer_info { // for returning cvMat as pyBuffer
                return py::buffer_info(
                    im.data,                                            // pointer to data
                    sizeof(unsigned char),                              // item size
                    py::format_descriptor<unsigned char>::format(),     // item descriptor
                    3,                                                  // matrix dimensionality
                    {                                                   // buffer dimensions
                        im.rows, 
                        im.cols, 
                        im.channels()
                    },          
                    {                                                    // strides in bytes
                        sizeof(unsigned char) * im.channels() * im.cols, // (issue with padding)
                        sizeof(unsigned char) * im.channels(),
                        sizeof(unsigned char)
                    }
                );
            })
        ;
        
}
