#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cmath>
#include <opencv2/core/cuda/common.hpp>
#include <opencv2/opencv.hpp>
#include <opencv2/cudev.hpp>
//#include <pybind11/pybind11.h>
//#include <pybind11/numpy.h>
#include <chrono>

// namespace py = pybind11;

typedef std::uint8_t imgScalar;
typedef double matScalar;

using namespace std;
using namespace cv;
 
cv::cuda::GpuMat H_d;

__global__ void undistortKernel
(
    cv::cuda::PtrStepSz<uchar3> img_d,
    cv::cuda::PtrStepSz<uchar3> img_u
)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int j = blockIdx.y * blockDim.y + threadIdx.y;

    img_u.ptr(i)[j] = img_d.ptr(i)[j];

}
/*
Mat pointwiseUndistort( py::array_t<imgScalar>& pyImg_d, 
                        py::array_t<matScalar>& pyH, 
                        py::tuple img_u_shape ){

    // --- Input data preparation --------------------------------------
     
    // link pyImg_d data to cv::Mat object img
    Mat img_d(
        pyImg_d.shape(0),               // rows
        pyImg_d.shape(1),               // cols
        CV_8UC3,                        // data type
        (imgScalar*)pyImg_d.data());    // data pointer
    
    cuda::GpuMat img_d_gpu(img_d); // create GpuMat from regular Mat

    // link H data to cv::Mat object
    Mat H(
        pyH.shape(0),                   // rows
        pyH.shape(1),                   // cols
        CV_64FC1,                       // data type
        (matScalar*)pyH.data());        // data pointer

    int M = img_u_shape[0].cast<int>();
    int N = img_u_shape[1].cast<int>();

    Mat img_u(M, N, CV_8UC3); // prepare return image
    cuda::GpuMat img_u_gpu(img_u); // create GpuMat from regular Mat

    // ---  Algorithm --------------------------------------------------
*/
int main(){


    Mat H(3,3,CV_32FC1); 
    // Construct H matrix (later passed by python)
    H.at<float>(0,0) = 3.55082e-1;  H.at<float>(0,0) = 1.51274e-1; H.at<float>(0,0) = 4.8e+1;
    H.at<float>(0,0) = -4.27999e-1; H.at<float>(0,0) = 5.60277e-1; H.at<float>(0,0) = 3.85e+2;
    H.at<float>(0,0) = -2.72420e-4; H.at<float>(0,0) = -1.27368e-4; H.at<float>(0,0) = 1e+0;
    
    H_d.create(3,3,CV_32FC1); // allocates space on GPU
    H_d.upload(H);
    
    // prep input image and return image  
    Mat img = imread("/app/_img/chessboard_perspective.jpg", IMREAD_COLOR );
    cv::cuda::GpuMat src;
    
    Mat ret;
    cv::cuda::GpuMat dst(800, 800, CV_8UC3); // allocate sapce 
    
    

    // Host code 
    
    src.upload(img);
    
    const dim3 blockSize(16,16);
    const dim3 gridSize(cv::cudev::divUp(dst.cols, blockSize.x), 
                        cv::cudev::divUp(dst.rows, blockSize.y)); // ceil: maybe not all threads used -> handle in kernel function


    // -- Kernel Launch 1 (slow) ------------------------------------------------------- 

    src.upload(img);
    
    undistortKernel<<<gridSize, blockSize>>>(src, dst);
    hipDeviceSynchronize();
    
    dst.download(ret);

    // -- Kernel Launch 2 (fast) ------------------------------------------------------- 
    auto start = chrono::steady_clock::now();
    src.upload(img);
    
    undistortKernel<<<gridSize, blockSize>>>(src, dst);
    hipDeviceSynchronize();
    
    dst.download(ret);
    auto end = chrono::steady_clock::now();

    // --------------------------------------------------------------------------

    // show results
    cout << "Elapsed time in microseconds: "
        << chrono::duration_cast<chrono::microseconds>(end - start).count()
        << " µs" << endl;

    imshow("gpu image", ret);
    waitKey(0);

    // free space
    //hipFree(u);

    return 0;
}       
/*
PYBIND11_MODULE(cppmodule, m){
    m.def("pointwiseUndistort", &pointwiseUndistort, py::return_value_policy::automatic);
    m.doc() = "Docstring for pointwiseUndistort function";

    py::class_<cv::Mat>(m, "Mat", py::buffer_protocol()) 
        .def_buffer([](cv::Mat &im) -> py::buffer_info {
                return py::buffer_info(
                    im.data,                                            // pointer to data
                    sizeof(unsigned char),                              // item size
                    py::format_descriptor<unsigned char>::format(),     // item descriptor
                    3,                                                  // matrix dimensionality
                    {                                                   // buffer dimensions
                        im.rows, 
                        im.cols, 
                        im.channels()
                    },          
                    {                                                    // strides in bytes
                        sizeof(unsigned char) * im.channels() * im.cols, // (issue with padding)
                        sizeof(unsigned char) * im.channels(),
                        sizeof(unsigned char)
                    }
                );
            })
        ;
    }
*/